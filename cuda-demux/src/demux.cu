#include "hip/hip_runtime.h"
#include "demux.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <unordered_map>
#include <vector>
#include <string>

struct Read {
    std::string sequence;
    std::string quality;
};

__global__ void barcode_matching_kernel(const char* reads, const char* barcodes, int* matches, int num_reads, int num_barcodes, int barcode_length) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_reads) {
        int best_match = -1;
        int min_mismatches = barcode_length + 1;

        for (int i = 0; i < num_barcodes; ++i) {
            int mismatches = 0;
            for (int j = 0; j < barcode_length; ++j) {
                if (reads[idx * barcode_length + j] != barcodes[i * barcode_length + j]) {
                    mismatches++;
                }
            }
            if (mismatches < min_mismatches) {
                min_mismatches = mismatches;
                best_match = i;
            }
        }

        matches[idx] = (min_mismatches <= 1) ? best_match : -1;
    }
}

std::unordered_map<std::string, std::vector<Read>> demux(const std::vector<Read>& reads, const std::string& samplesheet) {
    std::unordered_map<std::string, std::vector<Read>> demuxed_data;

    auto barcodes = load_barcodes(samplesheet);
    int num_reads = reads.size();
    int num_barcodes = barcodes.size();
    int barcode_length = barcodes[0].length();

    char* d_reads;
    char* d_barcodes;
    int* d_matches;

    hipMalloc(&d_reads, num_reads * barcode_length * sizeof(char));
    hipMalloc(&d_barcodes, num_barcodes * barcode_length * sizeof(char));
    hipMalloc(&d_matches, num_reads * sizeof(int));

    hipMemcpy(d_reads, reads.data(), num_reads * barcode_length * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_barcodes, barcodes.data(), num_barcodes * barcode_length * sizeof(char), hipMemcpyHostToDevice);

    int threads_per_block = 256;
    int blocks_per_grid = (num_reads + threads_per_block - 1) / threads_per_block;

    barcode_matching_kernel<<<blocks_per_grid, threads_per_block>>>(d_reads, d_barcodes, d_matches, num_reads, num_barcodes, barcode_length);

    std::vector<int> matches(num_reads);
    hipMemcpy(matches.data(), d_matches, num_reads * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < num_reads; ++i) {
        if (matches[i] != -1) {
            demuxed_data[barcodes[matches[i]]].push_back(reads[i]);
        }
    }

    hipFree(d_reads);
    hipFree(d_barcodes);
    hipFree(d_matches);

    return demuxed_data;
}